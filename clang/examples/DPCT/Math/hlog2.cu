// Option: --use-dpcpp-extensions=intel_device_math
// Option: --use-experimental-features=bfloat16_math_functions

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h, __hip_bfloat16 b) {
  // Start
  hlog2(h /*__half*/);
  hlog2(b /*__nv_bfloat16*/);
  // End
}
