// RUN: dpct --enable-profiling  -out-root %T/driver-stream-and-event-enable-profiling %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-stream-and-event-enable-profiling/driver-stream-and-event-enable-profiling.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/driver-stream-and-event-enable-profiling/driver-stream-and-event-enable-profiling.dp.cpp -o %T/driver-stream-and-event-enable-profiling/driver-stream-and-event-enable-profiling.dp.o %}

// CHECK:#define DPCT_PROFILING_ENABLED
// CHECK-NEXT:#include <sycl/sycl.hpp>
// CHECK-NEXT:#include <dpct/dpct.hpp>
// CHECK-NEXT:#include <vector>
#include "hip/hip_runtime.h"
#include <vector>

template <typename T>
void my_error_checker(T ReturnValue, char const *const FuncName) {
}

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)

void foo(){
  hipFunction_t f;
  hipStream_t s;
  hipEvent_t e;

// CHECK: s->ext_oneapi_submit_barrier({*e});
  hipEventCreateWithFlags(&e, hipEventDefault);
  hipStreamWaitEvent(s, e, 0);

// CHECK: *e = s->ext_oneapi_submit_barrier();
// CHECK-NEXT: e->wait_and_throw();
  hipEventRecord(e, s);
  hipEventSynchronize(e);

// CHECK: sycl::info::event_command_status r;
// CHECK-NEXT: r = e->get_info<sycl::info::event::command_execution_status>();
  hipError_t r;
  r = hipEventQuery(e);

// CHECK: dpct::event_ptr start, end;
// CHECK: *start = s->ext_oneapi_submit_barrier();
// CHECK: *end = s->ext_oneapi_submit_barrier();
// CHECK: start->wait_and_throw();
// CHECK: end->wait_and_throw();
// CHECK: float result_time;
// CHECK: result_time = (end->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
  hipEvent_t start, end;
  hipEventRecord(start, s);
  hipEventRecord(end, s);
  hipEventSynchronize(start);
  hipEventSynchronize(end);
  float result_time;
  hipEventElapsedTime(&result_time, start, end);
}
