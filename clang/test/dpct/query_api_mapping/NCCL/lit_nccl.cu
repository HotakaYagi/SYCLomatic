
#include <hip/hip_runtime.h>
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclGetVersion | FileCheck %s -check-prefix=ncclGetVersion
// ncclGetVersion: CUDA API:
// ncclGetVersion-NEXT:   ncclGetVersion(version /*int **/);
// ncclGetVersion-NEXT: Is migrated to:
// ncclGetVersion-NEXT:   *version = dpct::ccl::get_version();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclGetUniqueId | FileCheck %s -check-prefix=ncclGetUniqueId
// ncclGetUniqueId: CUDA API:
// ncclGetUniqueId-NEXT:   ncclGetUniqueId(uniqueId /*ncclUniqueId **/);
// ncclGetUniqueId-NEXT: Is migrated to:
// ncclGetUniqueId-NEXT:   *uniqueId = dpct::ccl::create_kvs_address();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommInitRank | FileCheck %s -check-prefix=ncclCommInitRank
// ncclCommInitRank: CUDA API:
// ncclCommInitRank-NEXT:   ncclCommInitRank(comm /*ncclComm_t **/, nranks /*int*/,
// ncclCommInitRank-NEXT:                   commId /*ncclUniqueId*/, rank /*int*/);
// ncclCommInitRank-NEXT: Is migrated to:
// ncclCommInitRank-NEXT:   *comm = new dpct::ccl::communicator_wrapper(nranks, rank, commId);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommDestroy | FileCheck %s -check-prefix=ncclCommDestroy
// ncclCommDestroy: CUDA API:
// ncclCommDestroy-NEXT:   ncclCommDestroy(comm /*ncclComm_t*/);
// ncclCommDestroy-NEXT: Is migrated to:
// ncclCommDestroy-NEXT:   delete comm;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommCount | FileCheck %s -check-prefix=ncclCommCount
// ncclCommCount: CUDA API:
// ncclCommCount-NEXT:    ncclCommCount(comm /*ncclComm_t*/, count /*int **/);
// ncclCommCount-NEXT: Is migrated to:
// ncclCommCount-NEXT:   *count = comm->size();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommCuDevice | FileCheck %s -check-prefix=ncclCommCuDevice
// ncclCommCuDevice: CUDA API:
// ncclCommCuDevice-NEXT:   ncclCommCuDevice(comm /*ncclComm_t*/, device /*int **/);
// ncclCommCuDevice-NEXT: Is migrated to:
// ncclCommCuDevice-NEXT:   *device = dpct::get_device_id(comm->get_device());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommUserRank | FileCheck %s -check-prefix=ncclCommUserRank
// ncclCommUserRank: CUDA API:
// ncclCommUserRank-NEXT:   ncclCommUserRank(comm /*ncclComm_t*/, rank /*int **/);
// ncclCommUserRank-NEXT: Is migrated to:
// ncclCommUserRank-NEXT:   *rank = comm->rank();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclAllReduce | FileCheck %s -check-prefix=ncclAllReduce
// ncclAllReduce: CUDA API:
// ncclAllReduce-NEXT:   ncclAllReduce(sendbuff /*void **/, recvbuff /*void **/, count /*size_t*/,
// ncclAllReduce-NEXT:                 datatype /*ncclDataType_t*/, op /*ncclRedOp_t*/,
// ncclAllReduce-NEXT:                 comm /*ncclComm_t*/, stream /*cudaStream_t*/);
// ncclAllReduce-NEXT: Is migrated to:
// ncclAllReduce-NEXT:   comm->allreduce(sendbuff, recvbuff, count, datatype, op, stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclBroadcast | FileCheck %s -check-prefix=ncclBroadcast
// ncclBroadcast: CUDA API:
// ncclBroadcast-NEXT:   ncclBroadcast(sendbuff /*void **/, recvbuff /*void **/, count /*size_t*/,
// ncclBroadcast-NEXT:             datatype /*ncclDataType_t*/, root /*int*/, comm /*ncclComm_t*/,
// ncclBroadcast-NEXT:             stream /*cudaStream_t*/);
// ncclBroadcast-NEXT: Is migrated to:
// ncclBroadcast-NEXT:   comm->broadcast(sendbuff, recvbuff, count, datatype, root, stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclReduce | FileCheck %s -check-prefix=ncclReduce
// ncclReduce: CUDA API:
// ncclReduce-NEXT:   ncclReduce(sendbuff /*void **/, recvbuff /*void **/, count /*size_t*/,
// ncclReduce-NEXT:             datatype /*ncclDataType_t*/, op /*ncclRedOp_t*/, root /*int*/,
// ncclReduce-NEXT:             comm /*ncclComm_t*/, stream /*cudaStream_t*/);
// ncclReduce-NEXT: Is migrated to:
// ncclReduce-NEXT:   comm->reduce(sendbuff, recvbuff, count, datatype, op, root, stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclReduceScatter | FileCheck %s -check-prefix=ncclReduceScatter
// ncclReduceScatter: CUDA API:
// ncclReduceScatter-NEXT:   ncclReduceScatter(sendbuff /*void **/, recvbuff /*void **/,
// ncclReduceScatter-NEXT:                     recvcount /*size_t*/, datatype /*ncclDataType_t*/,
// ncclReduceScatter-NEXT:                     op /*ncclRedOp_t*/, comm /*ncclComm_t*/,
// ncclReduceScatter-NEXT:                     stream /*cudaStream_t*/);
// ncclReduceScatter-NEXT: Is migrated to:
// ncclReduceScatter-NEXT:   comm->reduce_scatter(sendbuff, recvbuff, recvcount, datatype, op, stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclBcast | FileCheck %s -check-prefix=ncclBcast
// ncclBcast: CUDA API:
// ncclBcast-NEXT:   ncclBcast(buff /*void **/, count /*size_t*/, datatype /*ncclDataType_t*/,
// ncclBcast-NEXT:                 root /*int*/, comm /*ncclComm_t*/, stream /*cudaStream_t*/);
// ncclBcast-NEXT: Is migrated to:
// ncclBcast-NEXT:   comm->broadcast(buff, buff, count, datatype, root, stream);
