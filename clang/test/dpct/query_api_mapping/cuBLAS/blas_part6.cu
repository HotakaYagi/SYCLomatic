
#include <hip/hip_runtime.h>
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCsscal | FileCheck %s -check-prefix=cublasCsscal
// cublasCsscal: CUDA API:
// cublasCsscal-NEXT:   cublasCsscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const float **/,
// cublasCsscal-NEXT:                x /*cuComplex **/, incx /*int*/);
// cublasCsscal-NEXT: Is migrated to:
// cublasCsscal-NEXT:   oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha, *handle), (std::complex<float>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasIdamin | FileCheck %s -check-prefix=cublasIdamin
// cublasIdamin: CUDA API:
// cublasIdamin-NEXT:   cublasIdamin(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
// cublasIdamin-NEXT:                incx /*int*/, res /*int **/);
// cublasIdamin-NEXT: Is migrated to (with the option --no-dry-pattern):
// cublasIdamin-NEXT:   int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_in_order_queue());
// cublasIdamin-NEXT:   oneapi::mkl::blas::column_major::iamin(*handle, n, x, incx, res_temp_ptr_ct{{[0-9]+}}, oneapi::mkl::index_base::one).wait();
// cublasIdamin-NEXT:   int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
// cublasIdamin-NEXT:   dpct::dpct_memcpy(res, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
// cublasIdamin-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_in_order_queue());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSetMatrix | FileCheck %s -check-prefix=cublasSetMatrix
// cublasSetMatrix: CUDA API:
// cublasSetMatrix-NEXT:   cublasSetMatrix(rows /*int*/, cols /*int*/, elementsize /*int*/,
// cublasSetMatrix-NEXT:                   a /*const void **/, lda /*int*/, b /*void **/, ldb /*int*/);
// cublasSetMatrix-NEXT: Is migrated to:
// cublasSetMatrix-NEXT:   dpct::matrix_mem_copy((void*)b, (void*)a, ldb, lda, rows, cols, elementsize);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZscal | FileCheck %s -check-prefix=cublasZscal
// cublasZscal: CUDA API:
// cublasZscal-NEXT:   cublasZscal(handle /*cublasHandle_t*/, n /*int*/,
// cublasZscal-NEXT:               alpha /*const cuDoubleComplex **/, x /*cuDoubleComplex **/,
// cublasZscal-NEXT:               incx /*int*/);
// cublasZscal-NEXT: Is migrated to:
// cublasZscal-NEXT:   oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha, *handle), (std::complex<double>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCtrmv | FileCheck %s -check-prefix=cublasCtrmv
// cublasCtrmv: CUDA API:
// cublasCtrmv-NEXT:   cublasCtrmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCtrmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasCtrmv-NEXT:               n /*int*/, a /*const cuComplex **/, lda /*int*/,
// cublasCtrmv-NEXT:               x /*cuComplex **/, incx /*int*/);
// cublasCtrmv-NEXT: Is migrated to:
// cublasCtrmv-NEXT:   oneapi::mkl::blas::column_major::trmv(*handle, upper_lower, trans, unit_nonunit, n, (std::complex<float>*)a, lda, (std::complex<float>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZrot | FileCheck %s -check-prefix=cublasZrot
// cublasZrot: CUDA API:
// cublasZrot-NEXT:   cublasZrot(handle /*cublasHandle_t*/, n /*int*/, x /*cuDoubleComplex **/,
// cublasZrot-NEXT:              incx /*int*/, y /*cuDoubleComplex **/, incy /*int*/,
// cublasZrot-NEXT:              c /*const double **/, s /*const cuDoubleComplex **/);
// cublasZrot-NEXT: Is migrated to:
// cublasZrot-NEXT:   dpct::rot(*handle, n, x, dpct::library_data_t::complex_double, incx, y, dpct::library_data_t::complex_double, incy, c, s, dpct::library_data_t::complex_double);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSetVector | FileCheck %s -check-prefix=cublasSetVector
// cublasSetVector: CUDA API:
// cublasSetVector-NEXT:   cublasSetVector(n /*int*/, elementsize /*int*/, x /*const void **/,
// cublasSetVector-NEXT:                   incx /*int*/, y /*void **/, incy /*int*/);
// cublasSetVector-NEXT: Is migrated to:
// cublasSetVector-NEXT:   dpct::matrix_mem_copy((void*)y, (void*)x, incy, incx, 1, n, elementsize);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSetStream | FileCheck %s -check-prefix=cublasSetStream
// cublasSetStream: CUDA API:
// cublasSetStream-NEXT:   cublasSetStream(handle /*cublasHandle_t*/, stream /*cudaStream_t*/);
// cublasSetStream-NEXT: Is migrated to:
// cublasSetStream-NEXT:   handle = stream;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZdgmm | FileCheck %s -check-prefix=cublasZdgmm
// cublasZdgmm: CUDA API:
// cublasZdgmm-NEXT:   cublasZdgmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
// cublasZdgmm-NEXT:               m /*int*/, n /*int*/, a /*const cuDoubleComplex **/, lda /*int*/,
// cublasZdgmm-NEXT:               x /*const cuDoubleComplex **/, incx /*int*/,
// cublasZdgmm-NEXT:               c /*cuDoubleComplex **/, ldc /*int*/);
// cublasZdgmm-NEXT: Is migrated to:
// cublasZdgmm-NEXT:   oneapi::mkl::blas::column_major::dgmm(*handle, left_right, m, n, (std::complex<double>*)a, lda, (std::complex<double>*)x, incx, (std::complex<double>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZdrot | FileCheck %s -check-prefix=cublasZdrot
// cublasZdrot: CUDA API:
// cublasZdrot-NEXT:   cublasZdrot(handle /*cublasHandle_t*/, n /*int*/, x /*cuDoubleComplex **/,
// cublasZdrot-NEXT:               incx /*int*/, y /*cuDoubleComplex **/, incy /*int*/,
// cublasZdrot-NEXT:               c /*const double **/, s /*const double **/);
// cublasZdrot-NEXT: Is migrated to:
// cublasZdrot-NEXT:   oneapi::mkl::blas::column_major::rot(*handle, n, (std::complex<double>*)x, incx, (std::complex<double>*)y, incy, dpct::get_value(c, *handle), dpct::get_value(s, *handle));

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZhpr | FileCheck %s -check-prefix=cublasZhpr
// cublasZhpr: CUDA API:
// cublasZhpr-NEXT:   cublasZhpr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZhpr-NEXT:              n /*int*/, alpha /*const double **/, x /*const cuDoubleComplex **/,
// cublasZhpr-NEXT:              incx /*int*/, a /*cuDoubleComplex **/);
// cublasZhpr-NEXT: Is migrated to:
// cublasZhpr-NEXT:   oneapi::mkl::blas::column_major::hpr(*handle, upper_lower, n, dpct::get_value(alpha, *handle), (std::complex<double>*)x, incx, (std::complex<double>*)a);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCtpmv | FileCheck %s -check-prefix=cublasCtpmv
// cublasCtpmv: CUDA API:
// cublasCtpmv-NEXT:   cublasCtpmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCtpmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasCtpmv-NEXT:               n /*int*/, a /*const cuComplex **/, x /*cuComplex **/,
// cublasCtpmv-NEXT:               incx /*int*/);
// cublasCtpmv-NEXT: Is migrated to:
// cublasCtpmv-NEXT:   oneapi::mkl::blas::column_major::tpmv(*handle, upper_lower, trans, unit_nonunit, n, (std::complex<float>*)a, (std::complex<float>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDtrmv | FileCheck %s -check-prefix=cublasDtrmv
// cublasDtrmv: CUDA API:
// cublasDtrmv-NEXT:   cublasDtrmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDtrmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasDtrmv-NEXT:               n /*int*/, a /*const double **/, lda /*int*/, x /*double **/,
// cublasDtrmv-NEXT:               incx /*int*/);
// cublasDtrmv-NEXT: Is migrated to:
// cublasDtrmv-NEXT:   oneapi::mkl::blas::column_major::trmv(*handle, upper_lower, trans, unit_nonunit, n, a, lda, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCtrsm | FileCheck %s -check-prefix=cublasCtrsm
// cublasCtrsm: CUDA API:
// cublasCtrsm-NEXT:   cublasCtrsm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
// cublasCtrsm-NEXT:               upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
// cublasCtrsm-NEXT:               unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
// cublasCtrsm-NEXT:               alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasCtrsm-NEXT:               b /*cuComplex **/, ldb /*int*/);
// cublasCtrsm-NEXT: Is migrated to:
// cublasCtrsm-NEXT:   oneapi::mkl::blas::column_major::trsm(*handle, left_right, upper_lower, transa, unit_diag, m, n, dpct::get_value(alpha, *handle), (std::complex<float>*)a, lda, (std::complex<float>*)b, ldb);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCsyrk | FileCheck %s -check-prefix=cublasCsyrk
// cublasCsyrk: CUDA API:
// cublasCsyrk-NEXT:   cublasCsyrk(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCsyrk-NEXT:               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasCsyrk-NEXT:               alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasCsyrk-NEXT:               beta /*const cuComplex **/, c /*cuComplex **/, ldc /*int*/);
// cublasCsyrk-NEXT: Is migrated to:
// cublasCsyrk-NEXT:   oneapi::mkl::blas::column_major::syrk(*handle, upper_lower, trans, n, k, dpct::get_value(alpha, *handle), (std::complex<float>*)a, lda, dpct::get_value(beta, *handle), (std::complex<float>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDsyr2 | FileCheck %s -check-prefix=cublasDsyr2
// cublasDsyr2: CUDA API:
// cublasDsyr2-NEXT:   cublasDsyr2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDsyr2-NEXT:               n /*int*/, alpha /*const double **/, x /*const double **/,
// cublasDsyr2-NEXT:               incx /*int*/, y /*const double **/, incy /*int*/, a /*double **/,
// cublasDsyr2-NEXT:               lda /*int*/);
// cublasDsyr2-NEXT: Is migrated to:
// cublasDsyr2-NEXT:   oneapi::mkl::blas::column_major::syr2(*handle, upper_lower, n, dpct::get_value(alpha, *handle), x, incx, y, incy, a, lda);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasIdamax | FileCheck %s -check-prefix=cublasIdamax
// cublasIdamax: CUDA API:
// cublasIdamax-NEXT:   cublasIdamax(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
// cublasIdamax-NEXT:                incx /*int*/, res /*int **/);
// cublasIdamax-NEXT: Is migrated to (with the option --no-dry-pattern):
// cublasIdamax-NEXT:   int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_in_order_queue());
// cublasIdamax-NEXT:   oneapi::mkl::blas::column_major::iamax(*handle, n, x, incx, res_temp_ptr_ct{{[0-9]+}}, oneapi::mkl::index_base::one).wait();
// cublasIdamax-NEXT:   int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
// cublasIdamax-NEXT:   dpct::dpct_memcpy(res, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
// cublasIdamax-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_in_order_queue());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZgemv | FileCheck %s -check-prefix=cublasZgemv
// cublasZgemv: CUDA API:
// cublasZgemv-NEXT:   cublasZgemv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
// cublasZgemv-NEXT:               n /*int*/, alpha /*const cuDoubleComplex **/,
// cublasZgemv-NEXT:               a /*const cuDoubleComplex **/, lda /*int*/,
// cublasZgemv-NEXT:               x /*const cuDoubleComplex **/, incx /*int*/,
// cublasZgemv-NEXT:               beta /*const cuDoubleComplex **/, y /*cuDoubleComplex **/,
// cublasZgemv-NEXT:               incy /*int*/);
// cublasZgemv-NEXT: Is migrated to:
// cublasZgemv-NEXT:   oneapi::mkl::blas::column_major::gemv(*handle, trans, m, n, dpct::get_value(alpha, *handle), (std::complex<double>*)a, lda, (std::complex<double>*)x, incx, dpct::get_value(beta, *handle), (std::complex<double>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasIsamin | FileCheck %s -check-prefix=cublasIsamin
// cublasIsamin: CUDA API:
// cublasIsamin-NEXT:   cublasIsamin(handle /*cublasHandle_t*/, n /*int*/, x /*const float **/,
// cublasIsamin-NEXT:                incx /*int*/, res /*int **/);
// cublasIsamin-NEXT: Is migrated to (with the option --no-dry-pattern):
// cublasIsamin-NEXT:   int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_in_order_queue());
// cublasIsamin-NEXT:   oneapi::mkl::blas::column_major::iamin(*handle, n, x, incx, res_temp_ptr_ct{{[0-9]+}}, oneapi::mkl::index_base::one).wait();
// cublasIsamin-NEXT:   int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
// cublasIsamin-NEXT:   dpct::dpct_memcpy(res, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
// cublasIsamin-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_in_order_queue());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZswap | FileCheck %s -check-prefix=cublasZswap
// cublasZswap: CUDA API:
// cublasZswap-NEXT:   cublasZswap(handle /*cublasHandle_t*/, n /*int*/, x /*cuDoubleComplex **/,
// cublasZswap-NEXT:               incx /*int*/, y /*cuDoubleComplex **/, incy /*int*/);
// cublasZswap-NEXT: Is migrated to:
// cublasZswap-NEXT:   oneapi::mkl::blas::column_major::swap(*handle, n, (std::complex<double>*)x, incx, (std::complex<double>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSspr | FileCheck %s -check-prefix=cublasSspr
// cublasSspr: CUDA API:
// cublasSspr-NEXT:   cublasSspr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasSspr-NEXT:              n /*int*/, alpha /*const float **/, x /*const float **/,
// cublasSspr-NEXT:              incx /*int*/, a /*float **/);
// cublasSspr-NEXT: Is migrated to:
// cublasSspr-NEXT:   oneapi::mkl::blas::column_major::spr(*handle, upper_lower, n, dpct::get_value(alpha, *handle), x, incx, a);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDdot | FileCheck %s -check-prefix=cublasDdot
// cublasDdot: CUDA API:
// cublasDdot-NEXT:   cublasDdot(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
// cublasDdot-NEXT:              incx /*int*/, y /*const double **/, incy /*int*/,
// cublasDdot-NEXT:              res /*double **/);
// cublasDdot-NEXT: Is migrated to:
// cublasDdot-NEXT:   double* res_temp_ptr_ct{{[0-9]+}} = res;
// cublasDdot-NEXT:   if(sycl::get_pointer_type(res, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(res, handle->get_context())!=sycl::usm::alloc::shared) {
// cublasDdot-NEXT:     res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, dpct::get_in_order_queue());
// cublasDdot-NEXT:   }
// cublasDdot-NEXT:   oneapi::mkl::blas::column_major::dot(*handle, n, x, incx, y, incy, res_temp_ptr_ct{{[0-9]+}});
// cublasDdot-NEXT:   if(sycl::get_pointer_type(res, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(res, handle->get_context())!=sycl::usm::alloc::shared) {
// cublasDdot-NEXT:     handle->wait();
// cublasDdot-NEXT:     *res = *res_temp_ptr_ct{{[0-9]+}};
// cublasDdot-NEXT:     sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_in_order_queue());
// cublasDdot-NEXT:   }

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCsymv | FileCheck %s -check-prefix=cublasCsymv
// cublasCsymv: CUDA API:
// cublasCsymv-NEXT:   cublasCsymv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCsymv-NEXT:               n /*int*/, alpha /*const cuComplex **/, a /*const cuComplex **/,
// cublasCsymv-NEXT:               lda /*int*/, x /*const cuComplex **/, incx /*int*/,
// cublasCsymv-NEXT:               beta /*const cuComplex **/, y /*cuComplex **/, incy /*int*/);
// cublasCsymv-NEXT: Is migrated to:
// cublasCsymv-NEXT:   oneapi::mkl::blas::column_major::symv(*handle, upper_lower, n, dpct::get_value(alpha, *handle), (std::complex<float>*)a, lda, (std::complex<float>*)x, incx, dpct::get_value(beta, *handle), (std::complex<float>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDspr2 | FileCheck %s -check-prefix=cublasDspr2
// cublasDspr2: CUDA API:
// cublasDspr2-NEXT:   cublasDspr2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDspr2-NEXT:               n /*int*/, alpha /*const double **/, x /*const double **/,
// cublasDspr2-NEXT:               incx /*int*/, y /*const double **/, incy /*int*/, a /*double **/);
// cublasDspr2-NEXT: Is migrated to:
// cublasDspr2-NEXT:   oneapi::mkl::blas::column_major::spr2(*handle, upper_lower, n, dpct::get_value(alpha, *handle), x, incx, y, incy, a);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZherk | FileCheck %s -check-prefix=cublasZherk
// cublasZherk: CUDA API:
// cublasZherk-NEXT:   cublasZherk(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZherk-NEXT:               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasZherk-NEXT:               alpha /*const double **/, a /*const cuDoubleComplex **/,
// cublasZherk-NEXT:               lda /*int*/, beta /*const double **/, c /*cuDoubleComplex **/,
// cublasZherk-NEXT:               ldc /*int*/);
// cublasZherk-NEXT: Is migrated to:
// cublasZherk-NEXT:   oneapi::mkl::blas::column_major::herk(*handle, upper_lower, trans, n, k, dpct::get_value(alpha, *handle), (std::complex<double>*)a, lda, dpct::get_value(beta, *handle), (std::complex<double>*)c, ldc);
