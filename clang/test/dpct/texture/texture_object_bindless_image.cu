
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none --use-experimental-features=bindless_images -out-root %T/texture/texture_object_bindless_image %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/texture/texture_object_bindless_image/texture_object_bindless_image.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST %T/texture/texture_object_bindless_image/texture_object_bindless_image.dp.cpp -o %T/texture/texture_object_bindless_image/texture_object_bindless_image.dp.o %}

// CHECK: void kernel(sycl::ext::oneapi::experimental::sampled_image_handle tex) {
__global__ void kernel(hipTextureObject_t tex) {
  int i;
  float j, k;
  // CHECK: sycl::ext::oneapi::experimental::read_image<sycl::short2>(tex, (float)i);
  tex1Dfetch<short2>(tex, i);
  // CHECK: sycl::ext::oneapi::experimental::read_image<sycl::short2>(tex, (float)i);
  tex1D<short2>(tex, i);
  // CHECK: i = sycl::ext::oneapi::experimental::read_image<int>(tex, (float)i);
  tex1D(&i, tex, i);
  // CHECK: sycl::ext::oneapi::experimental::read_image<sycl::short2>(tex, sycl::float2(j, k));
  tex2D<short2>(tex, j, k);
  // CHECK: i = sycl::ext::oneapi::experimental::read_image<int>(tex, sycl::float2(j, k));
  tex2D(&i, tex, j, k);
}

int main() {
  void *input;
  size_t w, h, sizeInBytes, w_offest_src, h_offest_src, w_offest_dest, h_offest_dest;
  unsigned int flag;
  hipExtent e;
  // CHECK: dpct::experimental::image_mem_ptr pArr, pArr_src;
  hipArray_t pArr, pArr_src;
  // TODO: need support.
  // cudaMipmappedArray_t pMipMapArr;
  // CHECK: dpct::image_channel desc;
  hipChannelFormatDesc desc;
  // CHECK: pArr = new sycl::ext::oneapi::experimental::image_mem(sycl::ext::oneapi::experimental::image_descriptor(e, desc.get_channel_order(), desc.get_channel_type()), q_ct1);
  hipMalloc3DArray(&pArr, &desc, e);
  // CHECK: pArr = new sycl::ext::oneapi::experimental::image_mem(sycl::ext::oneapi::experimental::image_descriptor({w, h}, desc.get_channel_order(), desc.get_channel_type()), q_ct1);
  hipMallocArray(&pArr, &desc, w, h);
  // CHECK: desc = dpct::experimental::get_channel(pArr);
  // CHECK-NEXT: e = pArr->get_range();
  // CHECK-NEXT: flag = 0;
  hipArrayGetInfo(&desc, &e, &flag, pArr);
  // CHECK: dpct::experimental::dpct_memcpy(pArr_src, w_offest_src, h_offest_src, pArr, w_offest_dest, h_offest_dest, w, h, q_ct1);
  hipMemcpy2DArrayToArray(pArr, w_offest_dest, h_offest_dest, pArr_src,
                           w_offest_src, h_offest_src, w, h,
                           hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(pArr, w_offest_src, h_offest_src, input, w, w, h, q_ct1);
  hipMemcpy2DFromArray(input, w, pArr, w_offest_src, h_offest_src, w, h,
                        hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::async_dpct_memcpy(pArr, w_offest_src, h_offest_src, input, w, w, h, q_ct1);
  hipMemcpy2DFromArrayAsync(input, w, pArr, w_offest_src, h_offest_src, w, h,
                             hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(input, pArr, w_offest_dest, h_offest_dest, w, w, h, q_ct1);
  hipMemcpy2DToArray(pArr, w_offest_dest, h_offest_dest, input, w, w, h,
                      hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::async_dpct_memcpy(input, pArr, w_offest_dest, h_offest_dest, w, w, h, q_ct1);
  hipMemcpy2DToArrayAsync(pArr, w_offest_dest, h_offest_dest, input, w, w, h,
                           hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(pArr_src, w_offest_src, h_offest_src, pArr, w_offest_dest, h_offest_dest, w * h, q_ct1);
  cudaMemcpyArrayToArray(pArr, w_offest_dest, h_offest_dest, pArr_src,
                         w_offest_src, h_offest_src, w * h,
                         hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(pArr, w_offest_src, h_offest_src, input, w * h, q_ct1);
  hipMemcpyFromArray(input, pArr, w_offest_src, h_offest_src, w * h,
                      hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::async_dpct_memcpy(pArr, w_offest_src, h_offest_src, input, w * h, q_ct1);
  cudaMemcpyFromArrayAsync(input, pArr, w_offest_src, h_offest_src, w * h,
                           hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(input, pArr, w_offest_dest, h_offest_dest, w * h, q_ct1);
  hipMemcpyToArray(pArr, w_offest_dest, h_offest_dest, input, w * h,
                    hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::async_dpct_memcpy(input, pArr, w_offest_dest, h_offest_dest, w * h, q_ct1);
  cudaMemcpyToArrayAsync(pArr, w_offest_dest, h_offest_dest, input, w * h,
                         hipMemcpyHostToDevice);

  // CHECK: dpct::image_data resDesc0, resDesc1, resDesc2, resDesc3, resDesc4;
  hipResourceDesc resDesc0, resDesc1, resDesc2, resDesc3, resDesc4;
  // CHECK: resDesc0.set_data_type(dpct::image_data_type::matrix);
  resDesc0.resType = hipResourceTypeArray;
  // CHECK: resDesc1.set_data_ptr(pArr);
  resDesc1.res.array.array = pArr;
  // TODO: need support.
  // resDesc0.resType = cudaResourceTypeMipmappedArray;
  // resDesc2.res.mipmap.mipmap = pMipMapArr;
  // CHECK: resDesc0.set_data_type(dpct::image_data_type::linear);
  resDesc0.resType = hipResourceTypeLinear;
  // CHECK: resDesc3.set_data_ptr(input);
  resDesc3.res.linear.devPtr = input;
  // CHECK: resDesc3.set_channel(desc);
  resDesc3.res.linear.desc = desc;
  // CHECK: resDesc3.set_x(sizeInBytes);
  resDesc3.res.linear.sizeInBytes = sizeInBytes;
  // CHECK: resDesc0.set_data_type(dpct::image_data_type::pitch);
  resDesc0.resType = hipResourceTypePitch2D;
  // CHECK: resDesc4.set_data_ptr(input);
  resDesc4.res.pitch2D.devPtr = input;
  // CHECK: resDesc4.set_channel(desc);
  resDesc4.res.pitch2D.desc = desc;
  // CHECK: resDesc4.set_x(w);
  resDesc4.res.pitch2D.width = w;
  // CHECK: resDesc4.set_y(h);
  resDesc4.res.pitch2D.height = h;
  // CHECK: resDesc4.set_pitch(sizeInBytes);
  resDesc4.res.pitch2D.pitchInBytes = sizeInBytes;
  {
    // CHECK: dpct::image_data resDesc;
    hipResourceDesc resDesc;
    // CHECK: resDesc.set_data(pArr);
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = pArr;
  }
  {
    // CHECK: dpct::image_data resDesc;
    hipResourceDesc resDesc;
    // CHECK: resDesc.set_data(input, sizeInBytes, desc);
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = input;
    resDesc.res.linear.desc = desc;
    resDesc.res.linear.sizeInBytes = sizeInBytes;
  }
  {
    // CHECK: dpct::image_data resDesc;
    hipResourceDesc resDesc;
    // CHECK: resDesc.set_data(input, w, h, sizeInBytes, desc);
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = input;
    resDesc.res.pitch2D.desc = desc;
    resDesc.res.pitch2D.width = w;
    resDesc.res.pitch2D.height = h;
    resDesc.res.pitch2D.pitchInBytes = sizeInBytes;
  }

  // CHECK: dpct::sampling_info texDesc1, texDesc2, texDesc3, texDesc4;
  hipTextureDesc texDesc1, texDesc2, texDesc3, texDesc4;
  // CHECK: texDesc1.set(sycl::addressing_mode::repeat);
  texDesc1.addressMode[0] = hipAddressModeWrap;
  // CHECK: texDesc2.set(sycl::addressing_mode::clamp_to_edge);
  texDesc2.addressMode[0] = hipAddressModeClamp;
  // CHECK: texDesc3.set(sycl::addressing_mode::mirrored_repeat);
  texDesc3.addressMode[0] = hipAddressModeMirror;
  // CHECK: texDesc4.set(sycl::addressing_mode::clamp);
  texDesc4.addressMode[0] = hipAddressModeBorder;
  // CHECK: texDesc1.set(sycl::filtering_mode::nearest);
  texDesc1.filterMode = hipFilterModePoint;
  // CHECK: texDesc2.set(sycl::filtering_mode::linear);
  texDesc2.filterMode = hipFilterModeLinear;
  // CHECK: texDesc3.set(sycl::coordinate_normalization_mode::unnormalized);
  texDesc3.normalizedCoords = 0;
  // CHECK: texDesc4.set(sycl::coordinate_normalization_mode::normalized);
  texDesc4.normalizedCoords = 1;

  // CHECK: sycl::ext::oneapi::experimental::sampled_image_handle tex;
  hipTextureObject_t tex;
  // CHECK: tex = dpct::experimental::create_bindless_image(resDesc1, texDesc1);
  hipCreateTextureObject(&tex, &resDesc1, &texDesc1, NULL);
  // CHECK: desc = dpct::experimental::get_channel(pArr);
  hipGetChannelDesc(&desc, pArr);
  // CHECK: resDesc1 = dpct::experimental::get_data(tex);
  hipGetTextureObjectResourceDesc(&resDesc1, tex);
  // CHECK: texDesc1 = dpct::experimental::get_sampling_info(tex);
  hipGetTextureObjectTextureDesc(&texDesc1, tex);
  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT: sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT: [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:   kernel(tex);
  // CHECK-NEXT: });
  kernel<<<1, 1>>>(tex);
  // CHECK: dpct::experimental::destroy_bindless_image(tex, q_ct1);
  hipDestroyTextureObject(tex);
  // CHECK: sycl::ext::oneapi::experimental::free_image_mem(pArr->get_handle(), sycl::ext::oneapi::experimental::image_type::standard, dpct::get_in_order_queue());
  hipFreeArray(pArr);
  return 0;
}
